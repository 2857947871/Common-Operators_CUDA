#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>


int main() {
    int deviceCount = 0;

    // 获取当前机器的GPU数量
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        printf("There are no available device(s) that support CUDA\n");
    } else {
        printf("Detected %d CUDA Capable device(s)\n", deviceCount);
    }

    for (int dev = 0; dev < deviceCount; ++dev) {
        
        hipSetDevice(dev);

        // 初始化当前device的属性获取对象
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        // 显存容量
        printf("Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                static_cast<float>(deviceProp.totalGlobalMem / 1048576.0f),
                (unsigned long long)deviceProp.totalGlobalMem);

        // 时钟频率
        printf(" GPU Max Clock rate:                           %.0f MHz (%0.2f" "GHz)\n",
                deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);

        // L2 cache大小
        printf("L2 Cache Size:                                 %d bytes\n", deviceProp.l2CacheSize);

        // high-frequent used
        // 注释见每个printf内的字符串
        printf("Total amount of shared memory per block:       %zu bytes\n", deviceProp.sharedMemPerBlock);
        printf("Total shared memory per multiprocessor:        %zu bytes\n", deviceProp.sharedMemPerMultiprocessor);
        printf("Total number of registers available per block: %d\n", deviceProp.regsPerBlock);
        printf("Warp size:                                     %d\n", deviceProp.warpSize);
        printf("Maximum number of threads per multiprocessor:  %d\n", deviceProp.maxThreadsPerMultiProcessor);
        printf("Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
        printf("Max dimension size of a block size (x,y,z):   (%d, %d, %d)\n",
                deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
                deviceProp.maxThreadsDim[2]);
        printf("Max dimension size of a grid size (x,y,z):    (%d, %d, %d)\n",
                deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
                deviceProp.maxGridSize[2]);
    }


    return 0;
    }