# include "gemv.cuh"


// [1, N] * [N, M]
// notes: CPU res sometimes are trash values, which very weird, so I check result by printing each res skipping comparison with CPU res
// when compile to executable file named gemv, we can run by typing "./gemv 1" to run fp32 gemv and "./gemv" to run fp16 gemv
template <typename T>
void gemvCPU(T *mat, T *vec, float *dst, int M, int N) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            dst[i] += (float)vec[j] * (float)mat[i + j * M];
        }
        if (i < 5) {
            printf("cpu res = %f\n", dst[i]);
        }
    }
}

template <typename T>
bool CheckResult(T *out, float *groudtruth, int M) {
    for (int i = 0; i < M; i++) {
        printf("%d th comparsion: %f and %f \n", i, (float)out[i], groudtruth[i]);
    }
    return true;
}


// vec.shape = [1, N]
// mat.shape = [N, M] and matrix is row major order in memory
// 为什么在编译期间获取threads_per_value(处理M个数据需要多少thread): 
//  1. 我们可以提前知道
//  2. 方便写cuda kernel
//      x = threads / threads_per_value: 一个block可以处理 x 行
//      N / x: 一共需要多少个block, 或者一个block需要循环几次
# define GEMV_KERNEL(dtype)                                                                                                 \
    dtype *d_vec;                                                                                                           \
    dtype *d_mat;                                                                                                           \
    dtype *d_dst;                                                                                                           \
    constexpr int N = 256;                                                                                                  \
    constexpr int M = 256;                                                                                                  \
    dtype *vec = (dtype *)malloc(N * sizeof(dtype));                                                                        \
    hipMalloc((void **)&d_vec, N * sizeof(dtype));                                                                         \
    dtype *mat = (dtype *)malloc(M * N * sizeof(dtype));                                                                    \
    hipMalloc((void **)&d_mat, M *N * sizeof(dtype));                                                                      \
    dtype *dst = (dtype *)malloc(M * sizeof(dtype));                                                                        \
    hipMalloc((void **)&d_dst, M * sizeof(dtype));                                                                         \
    for (int i = 0; i < N; i++)                                                                                             \
    {                                                                                                                               \
        vec[i] = (dtype)1;                                                                                                          \
    }                                                                                                                               \
    for (int i = 0; i < N * M; i++)                                                                                                 \
    {                                                                                                                               \
        mat[i] = (dtype)1;                                                                                                          \
    }                                                                                                                               \
    hipMemcpy(d_vec, vec, N * sizeof(dtype), hipMemcpyHostToDevice);                                                              \
    hipMemcpy(d_mat, mat, M *N * sizeof(dtype), hipMemcpyHostToDevice);                                                           \
    constexpr int THREADS_PER_BLOCK = 256;                                                                                          \
    constexpr int VEC_SIZE = Vec<dtype>::size;                                                                                      \
    constexpr int THREADS_PER_VALUE = VecMat::get_thread_per_mat_row<M, dtype>::value;                                              \
    VecMat::DispatchLauncher<THREADS_PER_BLOCK, THREADS_PER_VALUE, VEC_SIZE>::template launcher<dtype>(d_mat, d_vec, d_dst, M, N);  \
    hipMemcpy(dst, d_dst, M * sizeof(dtype), hipMemcpyDeviceToHost);                                                              \
    float *groudtruth = (float *)malloc(sizeof(float) * M);                                                                         \
    gemvCPU(mat, vec, groudtruth, M, N);                                                                                    \
    bool is_right = CheckResult(dst, groudtruth, M);                                                                        \
    if (is_right)                                                                                                           \
    {                                                                                                                       \
        printf("the ans is right\n");                                                                                       \
    }                                                                                                                       \
    else                                                                                                                    \
    {                                                                                                                       \
        printf("the ans is wrong\n");                                                                                       \
    }                                                                                                                       \
    hipFree(d_vec);                                                                                                        \
    hipFree(d_mat);                                                                                                        \
    hipFree(d_dst);                                                                                                        \
    free(vec);                                                                                                              \
    free(mat);                                                                                                              \
    free(dst);


int main(int argc, char** argv)
{

    GEMV_KERNEL(float);


    return 0;
}