// reduce 类算子 -> 累加
// v2: 消除 shared memory 的 bank conflict
// baseline:  519.084167 ms
// reduce_v0: 0.696928 ms      bank conflict: 834
// reduce_v1: 0.510880 ms(old) bank conflict: 7001056
// reduce_v1: 0.478528 ms(new) bank conflict: 689
// reduce_v2: 0.454912 ms      bank conflict: 891


#include <hip/hip_runtime.h>


#include <bits/stdc++.h>

void CheckResult(int *out, int groudtruth, int n) {
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        printf("groudtruth: %d\n", groudtruth);
        printf("result: %d\n", res);
        printf("the ans is wrong\n");
    }

    else
        printf("the ans is right\n");
}

// blockSize 作为模板参数的效果主要是用于静态 shared memory 的申请, 需要传入编译期(间)常量指定大小
template<int blockSize>
__global__ void reduce_v2(int* input, int* output, size_t n) {

    int tid  = threadIdx.x;
    int gtid = blockIdx.x * blockSize + threadIdx.x;

    // load
    __shared__ float smem[blockSize];
    smem[tid] = input[gtid];
    __syncthreads(); // 确保所有 thread 完成 shared memory 的写入

    // operation
    // bank conflict
    // V1(旧版):
    // tid0: smem[0] 和 [1]
    // tid1: smem[2] 和 [3]
    // tid16: smem[32] 和 smem[33] -> conflict
    // V2:
    // 见笔记
    for (int idx = blockDim.x / 2; idx > 0; idx >>= 1) {
        if (tid < idx) {
            smem[tid] += smem[tid + idx];
        } 
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = smem[0];
    }
}


int main() {

    // 初始化变量
    float ms = 0;
    int groudtruth = 0;
    const int N = 25600000; // 数据量

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    const int blockSize = 256;
    // 向上加 1 -> 防止 N = 255
    int gridSize  = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]); 

    dim3 Grid(gridSize);
    dim3 Block(blockSize);

    // allocate memory
    int* device_in;
    int* device_out;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_out = (int* )malloc(gridSize * sizeof(int));
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        groudtruth += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_out, gridSize * sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // grid: (100000, 1, 1)   block: (256, 1, 1)   thread = 25600000
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v2<blockSize><<<Grid, Block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_out, device_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    CheckResult(host_out, groudtruth, gridSize);

    printf("reducev2 latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);


    return 0;
}