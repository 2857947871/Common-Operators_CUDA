// reduce 类算子 -> 累加
// v1: 改进 v0 的 %, * 运算, 使用位运算来代替
// baseline:  519.084167 ms
// reduce_v0: 0.696928 ms      bank conflict: 834
// reduce_v1: 0.510880 ms(old) bank conflict: 7001056
// reduce_v1: 0.478528 ms(new) bank conflict: 674


#include <hip/hip_runtime.h>


#include <bits/stdc++.h>

// blockSize 作为模板参数的效果主要是用于静态 shared memory 的申请, 需要传入编译期(间)常量指定大小
template<int blockSize>
__global__ void reduce_v1(int* input, int* output, size_t n) {

    int tid  = threadIdx.x;
    int gtid = blockIdx.x * blockSize + threadIdx.x;

    // load
    __shared__ float smem[blockSize];
    smem[tid] = input[gtid];
    __syncthreads(); // 确保所有 thread 完成 shared memory 的写入

    // operation
    for (int idx = 1; idx < blockDim.x; idx <<= 1) {
        if ((tid & (2 * idx - 1)) == 0) {
            smem[tid] += smem[tid + idx];
        } 
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = smem[0];
    }
}

void CheckResult(int *out, int groudtruth, int n) {
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        printf("groudtruth: %d\n", groudtruth);
        printf("result: %d\n", res);
        printf("the ans is wrong\n");
    }

    printf("the ans is right\n");
}

int main() {

    // 初始化变量
    float ms = 0;
    int groudtruth = 0;
    const int N = 25600000; // 数据量

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    const int blockSize = 256;
    // 向上加 1 -> 防止 N = 255
    int gridSize  = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]); 

    dim3 Grid(gridSize);
    dim3 Block(blockSize);

    // allocate memory
    int* device_in;
    int* device_out;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_out = (int* )malloc(gridSize * sizeof(int));
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        groudtruth += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_out, gridSize * sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // grid: (100000, 1, 1)   block: (256, 1, 1)   thread = 25600000
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v1<blockSize><<<Grid, Block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_out, device_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    CheckResult(host_out, groudtruth, gridSize);

    printf("reducev1 latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);


    return 0;
}