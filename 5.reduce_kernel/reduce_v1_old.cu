#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


// v1新版本: 用位运算替换除余操作
// latency: 2.825ms
// blockSize作为模板参数的效果主要用于静态shared memory的申请需要传入编译期常量指定大小（L120)
template<int blockSize>
__global__ void reduce_v1(float *d_in,float *d_out) {
     __shared__ float smem[blockSize];

     unsigned int tid = threadIdx.x;

     unsigned int gtid = blockIdx.x * blockSize + threadIdx.x;

     smem[tid] = d_in[gtid];

     __syncthreads();

     for(unsigned int s = 1; s < blockDim.x; s *= 2) {
         int index = 2 * s * tid;
         if (index < blockDim.x) {
             smem[index] += smem[index + s];
         }
         __syncthreads();
     }

     if (tid == 0) {
        d_out[blockIdx.x] = smem[0];
    }
}
bool CheckResult(float *out, float groudtruth, int n){
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {

        printf("res is: %f \n", res);
        return false;
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    //int GridSize = 100000;
    float *a = (float *)malloc(N * sizeof(float));
    float *d_a;
    hipMalloc((void **)&d_a, N * sizeof(float));

    float *out = (float*)malloc((GridSize) * sizeof(float));
    float *d_out;
    hipMalloc((void **)&d_out, (GridSize) * sizeof(float));

    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
    }

    float groudtruth = N * 1.0f;

    hipMemcpy(d_a, a, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v1<blockSize><<<Grid,Block>>>(d_a, d_out);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(out, d_out, GridSize * sizeof(float), hipMemcpyDeviceToHost);
    printf("allcated %d blocks, data counts are %d", GridSize, N);
    bool is_right = CheckResult(out, groudtruth, GridSize);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");

        printf("groudtruth is: %f \n", groudtruth);
    }
    printf("reduce_v1 latency = %f ms\n", milliseconds);

    hipFree(d_a);
    hipFree(d_out);
    free(a);
    free(out);
}

