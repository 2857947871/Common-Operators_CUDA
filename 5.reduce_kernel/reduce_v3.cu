// reduce 类算子 -> 累加
// v3: 空闲 thread(每个 stage 过后仅有前一个 stage 的一半在工作)也干活
// baseline:  519.084167 ms
// reduce_v0: 0.696928 ms      bank conflict: 834
// reduce_v1: 0.510880 ms(old) bank conflict: 7001056
// reduce_v1: 0.478528 ms(new) bank conflict: 689
// reduce_v2: 0.454912 ms      bank conflict: 891
// reduce_v3: 0.237376 ms      bank conflict: 987


#include <hip/hip_runtime.h>


#include <bits/stdc++.h>


void CheckResult(int *out, int groudtruth, int n) {
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        printf("groudtruth: %d\n", groudtruth);
        printf("result: %d\n", res);
        printf("the ans is wrong\n");
    }

    else
        printf("the ans is right\n");
}


template<int blockSize>
__global__ void reduce_v3(int* input, int* output, size_t n) {

    __shared__ float smem[blockSize];
    unsigned int tid = threadIdx.x;
    unsigned int gtid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // load
    smem[tid] = input[gtid] + input[gtid + blockSize];
    __syncthreads();

    // operation
    for (unsigned int idx = blockDim.x / 2; idx > 0; idx >>= 1) {
        if (tid < idx) {
            smem[tid] += smem[tid + idx];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = smem[0];
    }
}


int main() {

    // 初始化变量
    float ms = 0;
    int groudtruth = 0;
    const int N = 25600000; // 数据量

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    const int blockSize = 256;
    const int blockSize_v3 = blockSize / 2;
    int gridSize  = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]); // 向上加 1 -> 防止 N = 255 -> gridSize = 0

    dim3 Grid(gridSize);
    dim3 Block(blockSize_v3);

    // allocate memory
    int* device_in;
    int* device_out;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_out = (int* )malloc(gridSize * sizeof(int));
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        groudtruth += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_out, gridSize * sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // grid: (100000, 1, 1)   block: (256, 1, 1)   thread = 25600000
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v3<blockSize_v3><<<Grid, Block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_out, device_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    CheckResult(host_out, groudtruth, gridSize);

    printf("reducev2 latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);


    return 0;
}