// reduce 类算子 -> 累加
// baseline: CPU 的处理逻辑
// ms: 519.084167 ms
#include <hip/hip_runtime.h>


#include <bits/stdc++.h>


__global__ void reduce_baseline(int* input, int* output, size_t n) {

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    printf("tid: %d\n", idx);

    int sum = 0;
    for (int i = 0; i < n; ++i) {
        sum += input[i];
    }

    *output = sum;
}

bool CheckResult(int *out, int groudtruth) {
    if (*out != groudtruth) {
        return false;
    }
    return true;
}

int main() {

    // 初始化变量
    float ms = 0;
    int groudtruth = 0;
    const int N = 25600000; // 数据量

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    int gridSize  = 1;
    int blockSize = 1;
    dim3 Grid(gridSize);
    dim3 Block(blockSize);

    // allocate memory
    int* device_in;
    int* device_out;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_out = (int* )malloc(sizeof(int));
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        groudtruth += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_out, sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // CPU 的方式处理, 分配 1 个 thread
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_baseline<<<Grid, Block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_out, device_out, 1 * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    if (CheckResult(host_out, groudtruth)) {
        printf("the ans is right\n");
    }
    else {
        printf("groudtruth: %d\n", groudtruth);
        printf("result: %d\n", host_out);
        printf("the ans is wrong\n");
    }
    printf("reduce_baseline latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);


    return 0;
}