// reduce 类算子 -> 累加
// v5: 展开 for 循环
// baseline:  519.084167 ms
// reduce_v0: 0.696928 ms      bank conflict: 834
// reduce_v1: 0.510880 ms(old) bank conflict: 7001056
// reduce_v1: 0.478528 ms(new) bank conflict: 689
// reduce_v2: 0.454912 ms      bank conflict: 891
// reduce_v3: 0.237376 ms      bank conflict: 987
// reduce_v4: 0.197024 ms      bank conflict: 2343
// reduce_v5: 0.186400 ms


#include <hip/hip_runtime.h>

#include <bits/stdc++.h>

#define THREAD_PER_BLOCK 256
void CheckResult(int *out, int groudtruth, int n) {
    float res = 0;
    for (int i = 0; i < n; i++){
        res += out[i];
    }
    if (res != groudtruth) {
        printf("groudtruth: %d\n", groudtruth);
        printf("result: %d\n", res);
        printf("the ans is wrong\n");
    }

    else
        printf("the ans is right\n");
}

// __device__ GPU 端的函数, 编译器自行决定是否 inline
// 将 for 循环展开, 节省位运算
template <int blockSize>
__device__ void BlockSharedMemReduce(float* smem) {

    // for (unsigned int idx = blockDim.x / 2; idx > 0; idx >>= 1)
    //      if (tid < idx)
    //          smem[tid] += smem[tid + idx];
    if (blockSize >= 1024) {
        if (threadIdx.x < 512)
            smem[threadIdx.x] += smem[threadIdx.x + 512];
    }
    __syncthreads();
    if (blockSize >=512) {
        if (threadIdx.x < 256)
            smem[threadIdx.x] += smem[threadIdx.x + 256];
    }
    __syncthreads();
    if (blockSize >= 256) {
        if (threadIdx.x< 128)
            smem[threadIdx.x] += smem[threadIdx.x + 128];
    }
    __syncthreads();
    if (blockSize >= 128) {
        if (threadIdx.x < 64)
            smem[threadIdx.x] += smem[threadIdx.x + 64];
    }
    __syncthreads();

    // final warp
    if (threadIdx.x < 32) {
        volatile float* vshm = smem;
        if (blockDim.x >= 64)
            vshm[threadIdx.x] += vshm[threadIdx.x + 32];
        vshm[threadIdx.x] += vshm[threadIdx.x + 16];
        vshm[threadIdx.x] += vshm[threadIdx.x + 8];
        vshm[threadIdx.x] += vshm[threadIdx.x + 4];
        vshm[threadIdx.x] += vshm[threadIdx.x + 2]; 
        vshm[threadIdx.x] += vshm[threadIdx.x + 1];
    }
}

template<int blockSize>
__global__ void reduce_v5(int* input, int* output, size_t n) {

    // 初始化
    __shared__ float smem[THREAD_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int gtid = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // load
    smem[tid] = input[gtid] + input[gtid + blockDim.x];
    __syncthreads();

    //operation
    BlockSharedMemReduce<blockSize>(smem);

    // result
    if (tid == 0) {
        output[blockIdx.x] = smem[0];
    }
}


int main() {

    // 初始化变量
    float ms = 0;
    int groudtruth = 0;
    const int N = 25600000; // 数据量

    // 获取设备属性
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    // thread
    const int blockSize = 256;
    const int blockSize_v3 = blockSize / 2;
    int gridSize  = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]); // 向上加 1 -> 防止 N = 255 -> gridSize = 0

    dim3 Grid(gridSize);
    dim3 Block(blockSize_v3);

    // allocate memory
    int* device_in;
    int* device_out;
    int* host_in  = (int* )malloc(N * sizeof(int));
    int* host_out = (int* )malloc(gridSize * sizeof(int));
    for (int i = 0; i < N; ++i) {
        host_in[i] = 1;
        groudtruth += host_in[i];
    }
    hipMalloc((void** )&device_in, N * sizeof(int));
    hipMalloc((void** )&device_out, gridSize * sizeof(int));
    hipMemcpy(device_in, host_in, N * sizeof(int), hipMemcpyHostToDevice);

    // 开始处理
    // grid: (100000, 1, 1)   block: (256, 1, 1)   thread = 25600000
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    reduce_v5<blockSize_v3><<<Grid, Block>>>(device_in, device_out, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(host_out, device_out, gridSize * sizeof(int), hipMemcpyDeviceToHost);

    // 验证
    CheckResult(host_out, groudtruth, gridSize);

    printf("reducev5 latency = %f ms\n", ms);

    // free
    hipFree(device_in);
    hipFree(device_out);
    free(host_in);
    free(host_out);


    return 0;
}