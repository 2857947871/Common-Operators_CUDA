// 原子操作(串行): histogram latency = 1.269152 ms

# include <hip/hip_runtime.h>

# include <bits/stdc++.h>
__global__ void histgram(int *hist_data, int *bin_data) {
    
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    // error: 资源竞争
    // bin_data[hist_data[gtid]]++;
    // 原子加法, 并行 -> 串行
    atomicAdd(&bin_data[hist_data[gtid]], 1);
}

bool CheckResult(int *out, int* groudtruth, int N){
    for (int i = 0; i < N; i++){
        if (out[i] != groudtruth[i]) {
            return false;
        }
    }
    return true;
}


int main () {
    // init
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    float milliseconds = 0;
    const int N = 25600000;
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    dim3 Grid(GridSize);
    dim3 Block(blockSize);

    int* hist = (int* )malloc(N * sizeof(int));
    int* bin  = (int* )malloc(N * sizeof(int));
    int* bin_data;
    int* hist_data;
    hipMalloc((void **)&bin_data, 256 * sizeof(int));
    hipMalloc((void **)&hist_data, N * sizeof(int));

    for(int i = 0; i < N; i++){
        hist[i] = i % 256;
    }

    int *groudtruth = (int *)malloc(256 * sizeof(int));;
    for(int j = 0; j < 256; j++){
        groudtruth[j] = 100000;
    }
    hipMemcpy(hist_data, hist, N * sizeof(int), hipMemcpyHostToDevice);

    // operation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histgram<<<Grid, Block>>>(hist_data, bin_data);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(bin, groudtruth, 256);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < 256; i++){
            printf("%lf ", bin[i]);
        }
        printf("\n");
    }
    printf("histogram latency = %f ms\n", milliseconds);    

    // free
    hipFree(bin_data);
    hipFree(hist_data);
    free(bin);
    free(hist);
}