#include "hip/hip_runtime.h"
// 优化手段:
//  优化共享内存的使用
//  向量化的读写
// v0: 0.291008 ms
// v1: 0.287360 ms
// v2: 0.115872 ms

# include <hip/hip_runtime.h>
# include <iostream>
# include <bits/stdc++.h>
# include "hip/hip_runtime.h"
# include <>


// softmax公式
// e^(xi - max(xi)) / sigma(e^(xi - max(xi)))
void softmaxCPU(float* input, float* result, int rows, int cols){
    for (int j = 0; j < rows; j++) {
        float total = 0;
        float MAX = 0;

        // 找到最大值
        for(int i = 0; i < cols; i++) {
            MAX = max(input[j * cols + i], MAX);
        }

        // 计算total
        for(int i = 0; i < cols; i++) {
            total += exp(input[j * cols + i] - MAX);
        }

        // 计算softmax
        for(int i = 0; i < cols; i++) {
            result[j * cols + i] = exp(input[j * cols + i] - MAX) / total;
        }
    }
}

int CheckResult(float *out, float* groudtruth, int N){
    for (int i = 0; i < N; i++){
      if (abs(out[i] - groudtruth[i]) > 1e-5) {
          printf("i: %d\n", i);
          return i;
      }
    }

    return -1;
}

// 策略:
// 1000 * 1024
//  1000个block, 每个block有1024个thread
//  每个block也互不干扰(因为是不同的batch) -> 每个block都有自己的max_val和sum -> shared memory(每个block公用一个)
//  -> 选择每个block的第0个thread来计算当前blokc的max_val(shared memory)和sum(shared memory) -> 最后求解
// softmax:
//  y = e^(x - max(x)) / sigma(e^(x - max(x)))
template<int classSize,  typename T>
__global__ void softmax_gpu_v2(T* x, T* y, int N) {

    int tid = threadIdx.x; // 0 ~ 255
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid * 4 >= N) {
        return;
    }

    // 向量化加载数据到shared memory
    __shared__ T smem[classSize]; // 一个block有1024个thread

    smem[tid * 4 + 0] = x[gid * 4 + 0];
    smem[tid * 4 + 1] = x[gid * 4 + 1];
    smem[tid * 4 + 2] = x[gid * 4 + 2];
    smem[tid * 4 + 3] = x[gid * 4 + 3];
    __syncthreads();

    // 计算每个block(每个smem)的max_val
    __shared__ T max_val;
    max_val = 0;
    if (tid == 0) {
        for (int i = 0; i < classSize; ++i) {
            max_val = max(max_val, smem[i]);
        }
    }
    __syncthreads();

    // 计算每个block(每个smem)的sum
    __shared__ T sum;
    sum = 0;
    if (tid == 0) {
        for (int i = 0; i < classSize; ++i) {
            sum += exp(smem[i] - max_val);
        }
    }
    __syncthreads();

    // 向量化计算softmax
    y[gid * 4 + 0] = exp(smem[tid * 4 + 0] - max_val) / sum;
    y[gid * 4 + 1] = exp(smem[tid * 4 + 1] - max_val) / sum;
    y[gid * 4 + 2] = exp(smem[tid * 4 + 2] - max_val) / sum;
    y[gid * 4 + 3] = exp(smem[tid * 4 + 3] - max_val) / sum;

    __syncthreads();
}


int main() {

    // 初始化
    int N = 1000 * 1024; // 1000行(batch_xize) 1024列(种类)
    float* x_host = (float* )malloc(N * sizeof(float));
    float* y_host = (float* )malloc(N * sizeof(float));
    float* groundtruth = (float* )malloc(N * sizeof(float));

    for (int i = 0; i < N; ++i) {
        x_host[i] = i % 10;
        y_host[i] = 0;
        groundtruth[i] = 0;
    }

    float* x_device;
    float* y_device;
    hipMalloc((void** )&x_device, N * sizeof(float));
    hipMalloc((void** )&y_device, N * sizeof(float));
    hipMemcpy(x_device, x_host, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_device, y_host, N * sizeof(float), hipMemcpyHostToDevice);

    constexpr int classSize = 1024;                  // 1024个thread(种类)
    int gridSize = (N + classSize - 1) / classSize;  // 1000个block(batch_size)

    float ms = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    softmax_gpu_v2<classSize><<<gridSize, classSize / 4>>>(x_device, y_device, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ms, start, stop);
    hipMemcpy(y_host, y_device, N * sizeof(float), hipMemcpyDeviceToHost);

    // 验证
    softmaxCPU(x_host, groundtruth, 1000, 1024);
    int is_right = CheckResult(y_host, groundtruth, N);
    if(is_right == -1) {
        printf("the ans is right\n");
    } else {
        for(int i = is_right; i < is_right + 10; i++){ printf("%lf ", y_host[i]); }
        printf("\n");

        for(int i = is_right; i < is_right + 10; i++){ printf("%lf ", groundtruth[i]); }
        printf("\n");

        printf("the ans is wrong\n");
    }
    printf("Softmax_GPU: %f ms\n", ms);

    // free
    free(x_host);
    free(y_host);
    free(groundtruth);
    hipFree(x_device);
    hipFree(y_device);

    return 0;
}