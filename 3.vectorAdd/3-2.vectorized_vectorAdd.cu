#include <stdio.h>
#include <hip/hip_runtime.h>


#define ARRAY_SIZE 		100000000   //Array size has to exceed L2 size to avoid L2 cache residence
#define MEMORY_OFFSET 	10000000
#define BENCH_ITER 		10
#define THREADS_NUM 	256

__device__ __forceinline__
float4 LoadFromGlobalPTX(float4 *ptr) {
    float4 ret;
    // ptx指令，是CUDA的更底层的语言，类似于汇编对于C/C++
    asm volatile (
        "ld.global.v4.f32 {%0, %1, %2, %3}, [%4];"
        : "=f"(ret.x), "=f"(ret.y), "=f"(ret.z), "=f"(ret.w)
        : "l"(ptr)
    );

    return ret;
}

//float4 vectoradd
__global__ void mem_bw (float* A,  float* B, float* C){
    
    // 泛指当前线程在所有block范围内的全局id
    // Dim: 常量   Idx: 变量
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
    // 分配的 thread 小于数据量
    // 前几个 thread 干多点
	for(int i = idx; i < MEMORY_OFFSET / 4; i += blockDim.x * gridDim.x) {

		// 向量化的读取
		// float4: 4个float的向量
		// reinterpret_cast: 强制类型转换   flota* -> float4* 
		float4 a1 = reinterpret_cast<float4*>(A)[i];
		float4 b1 = reinterpret_cast<float4*>(B)[i];
		float4 c1;

		// 问题: 为什么不直接相加?
		// 结论: cuda 支持向量化操作的读和写, 但是计算还是要一个一个算(标量)
		// 测量显存带宽方法1:向量加法,248.8g/s
		// 标量化的计算
		c1.x = a1.x + b1.x;
		c1.y = a1.y + b1.y;
		c1.z = a1.z + b1.z;
		c1.w = a1.w + b1.w;

		// 向量化的写入
		reinterpret_cast<float4*>(C)[i] = c1;
	}
}

void vec_add_cpu(float *x, float *y, float *z, int N)
{
    for (int i = 0; i < 20; i++) z[i] = y[i] + x[i];
}

int main(){
	float *A = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *B = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *C = (float*) malloc(ARRAY_SIZE*sizeof(float));

	float *A_g;
	float *B_g;
	float *C_g;

	float milliseconds = 0;

	for (uint32_t i=0; i<ARRAY_SIZE; i++){
		A[i] = (float)i;
		B[i] = (float)i;
	}
	hipMalloc((void**)&A_g, ARRAY_SIZE*sizeof(float));
	hipMalloc((void**)&B_g, ARRAY_SIZE*sizeof(float));
	hipMalloc((void**)&C_g, ARRAY_SIZE*sizeof(float));

	hipMemcpy(A_g, A, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_g, B, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  
	int BlockNums = MEMORY_OFFSET / 256;

	//warm up to occupy L2 cache
	printf("warm up start\n");
	mem_bw<<<BlockNums / 4, THREADS_NUM>>>(A_g, B_g, C_g);
	printf("warm up end\n");

	// time start using cudaEvent
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	for (int i = BENCH_ITER - 1; i >= 0; --i) {
		mem_bw<<<BlockNums / 4, THREADS_NUM>>>(A_g + i * MEMORY_OFFSET, B_g + i * MEMORY_OFFSET, C_g + i * MEMORY_OFFSET);
	}
	
	// time stop using cudaEvent
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(C, C_g, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	/* CPU compute */
	float* C_cpu_res = (float *) malloc(20*sizeof(float));
	vec_add_cpu(A, B, C_cpu_res, ARRAY_SIZE);

	/* check GPU result with CPU*/
	for (int i = 0; i < 20; ++i) {
		/* 测量显存带宽时, 修改C_cpu_res[i]为0 */
		if (fabs(C_cpu_res[i] - C[i]) > 1e-6) {
			printf("Result verification failed at element index %d!\n", i);
		}
	}
	printf("Result right\n");
	unsigned N = ARRAY_SIZE * 4;
	
	/* 测量显存带宽时, 根据实际读写的数组个数, 指定下行是 1*(float)N 还是 2*(float)N 还是 3*(float)N */
	printf("Mem BW= %f (GB/sec)\n", 3 * (float)N / milliseconds / 1e6);
  	hipFree(A_g);
  	hipFree(B_g);
  	hipFree(C_g);

  	free(A);
  	free(B);
  	free(C);
  	free(C_cpu_res);
}
